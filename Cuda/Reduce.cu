#include "hip/hip_runtime.h"
// Gkyl ------------------------------------------------------------------------
//
// Functions to compute reductions in GPU (Cuda).
//
//    _______     ___
// + 6 @ |||| # P ||| +
//------------------------------------------------------------------------------

#include <Reduce.h>

bool isPow2(unsigned int x) { return ((x & (x - 1)) == 0); }

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

__device__ double redBinOpMax(double a, double b) {
  return MAX(a,b);
}
__device__ redBinOpFunc_t d_redBinOpMaxPtr = &redBinOpMax;

redBinOpFunc_t getRedBinOpFuncFromDevice(unsigned int redBinOpLabel) {
  redBinOpFunc_t redBinOpFuncPtr; 
  auto err = hipMemcpyFromSymbol(&redBinOpFuncPtr, HIP_SYMBOL(d_redBinOpMaxPtr), sizeof(redBinOpFunc_t));
  return redBinOpFuncPtr;
}

// Compute the number of threads and blocks to use for the given reduction
// kerne. We set threads/block to the minimum of maxThreads and n/2.
// We observe the maximum specified number of blocks, because
// each thread in the kernel can process a variable number of elements.
void reductionBlocksAndThreads(GkDeviceProp *prop, int numElements, int maxBlocks,
                               int maxThreads, int &blocks, int &threads) {

  threads = (numElements < maxThreads * 2) ? nextPow2((numElements + 1) / 2) : maxThreads;
  blocks  = (numElements + (threads * 2 - 1)) / (threads * 2);

  if ((float)threads * blocks >
      (float)(prop->maxGridSize)[0] * prop->maxThreadsPerBlock) {
    printf("n is too large, please choose a smaller number!\n");
  }

  if (blocks > (prop->maxGridSize)[0]) {
    printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
        blocks, (prop->maxGridSize)[0], threads * 2, threads);

    blocks  /= 2;
    threads *= 2;
  }

  blocks = MIN(maxBlocks, blocks);
}

// This algorithm reduces multiple elements per thread sequentially. This reduces
// the overall cost of the algorithm while keeping the work complexity O(n) and
// the step complexity O(log n). (Brent's Theorem optimization)
// Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
// In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
// If blockSize > 32, allocate blockSize*sizeof(T) bytes.
template <unsigned int BLOCKSIZE, bool nIsPow2>
__global__ void d_reduce(baseReduceOp *redOpIn, double *dataIn, double *out, unsigned int nElements) {
  // Handle to thread block group.
  cg::thread_block cgThreadBlock = cg::this_thread_block();
  extern __shared__ double sdata[];  // Stores partial reductions.

  // Perform first level of reduction, reading from global memory, writing to shared memory.
  unsigned int tID       = threadIdx.x;
  unsigned int linearIdx = blockIdx.x * BLOCKSIZE * 2 + threadIdx.x;
  unsigned int gridSize  = BLOCKSIZE * 2 * gridDim.x;

  double myReduc = redOpIn->initValue;

  // We reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (linearIdx < nElements) {
    myReduc = redOpIn->reduce(myReduc, dataIn[linearIdx]);

    // Ensure we don't read out of bounds (optimized away for powerOf2 sized arrays)/
    if (nIsPow2 || linearIdx+BLOCKSIZE<nElements) myReduc = redOpIn->reduce(myReduc, dataIn[linearIdx+BLOCKSIZE]);

    linearIdx += gridSize;
  }

  // Each thread puts its local reduction into shared memory.
  sdata[tID] = myReduc;
  cg::sync(cgThreadBlock);

  // Do reduction in shared mem.
  if ((BLOCKSIZE >= 512) && (tID < 256)) {
    sdata[tID] = myReduc = redOpIn->reduce(myReduc, sdata[tID + 256]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 256) && (tID < 128)) {
    sdata[tID] = myReduc = redOpIn->reduce(myReduc, sdata[tID + 128]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 128) && (tID < 64)) {
    sdata[tID] = myReduc = redOpIn->reduce(myReduc, sdata[tID + 64]);
  }

  cg::sync(cgThreadBlock);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cgThreadBlock);

  if (cgThreadBlock.thread_rank() < 32) {
    // Fetch final intermediate reduction from 2nd warp.
    if (BLOCKSIZE >= 64) myReduc = redOpIn->reduce(myReduc, sdata[tID + 32]);
    // Reduce final warp using shuffle.
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      double shflMax = tile32.shfl_down(myReduc, offset);
      myReduc = redOpIn->reduce(myReduc, shflMax);
    }
  }

  // Write result for this block to global mem.
  if (cgThreadBlock.thread_rank() == 0) { out[blockIdx.x] = myReduc; }
}

void reduceDeviceArray(baseReduceOp *opIn, int numElements, int blocks, int threads, double *d_dataIn, double *d_dataOut) {
  // Launch the device kernel that reduces a device array 'd_dataIn'
  // containing 'numElements' elements.

  // When there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

  if (isPow2(numElements)) {
    switch (threads) {
      case 512:
        d_reduce<512,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 256:
        d_reduce<256,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 128:
        d_reduce<128,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 64:
        d_reduce<64,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 32:
        d_reduce<32,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 16:
        d_reduce<16,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 8:
        d_reduce<8,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 4:
        d_reduce<4,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 2:
        d_reduce<2,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 1:
        d_reduce<1,true><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
    }
  } else {
    switch (threads) {
      case 512:
        d_reduce<512,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 256:
        d_reduce<256,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 128:
        d_reduce<128,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 64:
        d_reduce<64,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 32:
        d_reduce<32,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 16:
        d_reduce<16,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 8:
        d_reduce<8,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 4:
        d_reduce<4,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 2:
        d_reduce<2,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
      case 1:
        d_reduce<1,false><<<blocks,threads,smemSize>>>(opIn, d_dataIn, d_dataOut, numElements);
        break;
    }
  }
}

