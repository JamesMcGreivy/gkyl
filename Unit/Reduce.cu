#include "hip/hip_runtime.h"
// Gkyl ------------------------------------------------------------------------
//
// Functions to compute reductions in GPU (Cuda).
//
//    _______     ___
// + 6 @ |||| # P ||| +
//------------------------------------------------------------------------------

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <float.h>

#include <GkylCudaFuncs.h>
#include <GkylCartField.h>
#include <GkylRange.h>

namespace cg = cooperative_groups;

extern "C"
{
  void getNumBlocksAndThreads(GkDeviceProp *prop, int numElements, int maxBlocks,
                              int maxThreads, int &blocks, int &threads);
  void cuda_cartFieldReduce(const int reduceOp, int numCellsTot, int numBlocks, int numThreads, int maxBlocks, int maxThreads,
                    GkDeviceProp *prop, GkylCartField_t *fIn, double *blockOut, double *intermediate, double *out);
}

bool isPow2(unsigned int x) { return ((x & (x - 1)) == 0); }

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

#ifndef binOpMin
#define binOpMin 1
#endif

#ifndef binOpMax
#define binOpMax 2
#endif

#ifndef binOpSum
#define binOpSum 3
#endif

#ifndef MIN
#define MIN(x, y) ((x < y) ? x : y)
#endif

#ifndef MAX
#define MAX(x, y) ((x > y) ? x : y)
#endif

#ifndef SUM
#define SUM(x,y) (x+y)
#endif

// Compute the number of threads and blocks to use for the given reduction
// kerne. We set threads/block to the minimum of maxThreads and n/2.
// We observe the maximum specified number of blocks, because
// each thread in the kernel can process a variable number of elements.
void getNumBlocksAndThreads(GkDeviceProp *prop, int numElements, int maxBlocks,
                            int maxThreads, int &blocks, int &threads) {

  threads = (numElements < maxThreads * 2) ? nextPow2((numElements + 1) / 2) : maxThreads;
  blocks  = (numElements + (threads * 2 - 1)) / (threads * 2);

  if ((float)threads * blocks >
      (float)(prop->maxGridSize)[0] * prop->maxThreadsPerBlock) {
    printf("n is too large, please choose a smaller number!\n");
  }

  if (blocks > (prop->maxGridSize)[0]) {
    printf("Grid size <%d> exceeds the device capability <%d>, set block size as %d (original %d)\n",
        blocks, (prop->maxGridSize)[0], threads * 2, threads);

    blocks  /= 2;
    threads *= 2;
  }

  blocks = MIN(maxBlocks, blocks);
}

template <unsigned int binOpType>
__inline__ __device__ double binOp(double a, double b) {
  double result = 0.0;
  if (binOpType==binOpMin) {
    result = MIN(a,b);
  } else if (binOpType==binOpMax) {
    result = MAX(a,b);
  } else if (binOpType==binOpSum) {
    result = SUM(a,b);
  }
  return result;
}

// This version adds multiple elements per thread sequentially. This reduces
// the overall cost of the algorithm while keeping the work complexity O(n) and
// the step complexity O(log n). (Brent's Theorem optimization)
// Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
// In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
// If blockSize > 32, allocate blockSize*sizeof(T) bytes.
template <unsigned int BLOCKSIZE, bool nIsPow2, unsigned int binOpType>
__global__ void d_reduceCartField(GkylCartField_t *fIn, double *redPerBlock) {
  // Handle to thread block group.
  cg::thread_block cgThreadBlock = cg::this_thread_block();
  extern __shared__ double sdata[];  // Stores partial reductions.

  // Perform first level of reduction, reading from global memory, writing to shared memory.
  unsigned int tID       = threadIdx.x;
  unsigned int linearIdx = blockIdx.x * BLOCKSIZE * 2 + threadIdx.x;
  unsigned int gridSize  = BLOCKSIZE * 2 * gridDim.x;

  double myReduc = 0;
  if (binOpType==binOpMin) {
    myReduc = DBL_MAX;
  } else if (binOpType==binOpMax) {
    myReduc = -DBL_MAX;
  }

  GkylRange_t *localRange  = fIn->localRange;
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer fIdxr   = fIn->genIndexer();
  unsigned int numCellsTot = localRange->volume();

  // We reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim). More blocks will result
  // in a larger gridSize and therefore fewer elements per thread.
  while (linearIdx < numCellsTot) {
    int idx[6];  // Should be CDIM+VDIM, but onerous to template just for this.
    localIdxr.invIndex(linearIdx, idx);
    int linIdx        = fIdxr.index(idx);
    const double *fld = fIn->getDataPtrAt(linIdx);

    myReduc = binOp<binOpType>(myReduc, fld[0]);

    // Ensure we don't read out of bounds (optimized away for powerOf2 sized arrays).
    unsigned int newLinearIdx = linearIdx+BLOCKSIZE;
    if (nIsPow2 || newLinearIdx<numCellsTot) {
      localIdxr.invIndex(newLinearIdx, idx);
      linIdx = fIdxr.index(idx);
      fld    = fIn->getDataPtrAt(linIdx);

      myReduc = binOp<binOpType>(myReduc, fld[0]);
    }

    linearIdx += gridSize;
  }

  // Each thread puts its local max into shared memory.
  sdata[tID] = myReduc;
  cg::sync(cgThreadBlock);

  // Do reduction in shared mem.
  if ((BLOCKSIZE >= 512) && (tID < 256)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 256]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 256) && (tID < 128)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 128]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 128) && (tID < 64)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 64]);
  }

  cg::sync(cgThreadBlock);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cgThreadBlock);

  if (cgThreadBlock.thread_rank() < 32) {
    // Fetch final intermediate max from 2nd warp.
    if (BLOCKSIZE >= 64) myReduc = binOp<binOpType>(myReduc, sdata[tID + 32]);
    // Reduce final warp using shuffle.
    for (int offset = tile32.size()/2; offset > 0; offset /= 2) {
      double shflMax = tile32.shfl_down(myReduc, offset);
      myReduc = binOp<binOpType>(myReduc, shflMax);
    }
  }

  // Write result for this block to global mem.
  if (cgThreadBlock.thread_rank() == 0) { redPerBlock[blockIdx.x] = myReduc; }
}

// This algorithm reduces multiple elements per thread sequentially. This reduces
// the overall cost of the algorithm while keeping the work complexity O(n) and
// the step complexity O(log n). (Brent's Theorem optimization)
// Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
// In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
// If blockSize > 32, allocate blockSize*sizeof(T) bytes.
template <unsigned int BLOCKSIZE, bool nIsPow2, unsigned int binOpType>
__global__ void d_reduce(double *dataIn, double *out, unsigned int nElements) {
  // Handle to thread block group.
  cg::thread_block cgThreadBlock = cg::this_thread_block();
  extern __shared__ double sdata[];  // Stores partial reductions.

  // Perform first level of reduction, reading from global memory, writing to shared memory.
  unsigned int tID       = threadIdx.x;
  unsigned int linearIdx = blockIdx.x * BLOCKSIZE * 2 + threadIdx.x;
  unsigned int gridSize  = BLOCKSIZE * 2 * gridDim.x;

  double myReduc = 0.0;
  if (binOpType==binOpMin) {
    myReduc = DBL_MAX;
  } else if (binOpType==binOpMax) {
    myReduc = -DBL_MAX;
  }

  // We reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (linearIdx < nElements) {
    myReduc = binOp<binOpType>(myReduc, dataIn[linearIdx]);

    // Ensure we don't read out of bounds (optimized away for powerOf2 sized arrays)/
    if (nIsPow2 || linearIdx+BLOCKSIZE<nElements) myReduc = binOp<binOpType>(myReduc, dataIn[linearIdx+BLOCKSIZE]);

    linearIdx += gridSize;
  }

  // Each thread puts its local reduction into shared memory.
  sdata[tID] = myReduc;
  cg::sync(cgThreadBlock);

  // Do reduction in shared mem.
  if ((BLOCKSIZE >= 512) && (tID < 256)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 256]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 256) && (tID < 128)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 128]);
  }

  cg::sync(cgThreadBlock);

  if ((BLOCKSIZE >= 128) && (tID < 64)) {
    sdata[tID] = myReduc = binOp<binOpType>(myReduc, sdata[tID + 64]);
  }

  cg::sync(cgThreadBlock);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cgThreadBlock);

  if (cgThreadBlock.thread_rank() < 32) {
    // Fetch final intermediate reduction from 2nd warp.
    if (BLOCKSIZE >= 64) myReduc = binOp<binOpType>(myReduc, sdata[tID + 32]);
    // Reduce final warp using shuffle.
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      double shflMax = tile32.shfl_down(myReduc, offset);
      myReduc = binOp<binOpType>(myReduc, shflMax);
    }
  }

  // Write result for this block to global mem.
  if (cgThreadBlock.thread_rank() == 0) { out[blockIdx.x] = myReduc; }
}

void reduceCartField(int opIn, int numCellsTot, int blocks, int threads, GkylCartField_t *fIn, double *blockRed) {
  // Launch the device kernel that reduces a CartField to an array,
  // each element of the array being the reduction performed by a block.

  // When there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

  if (isPow2(numCellsTot)) {
    switch (threads) {
      case 512:
        switch (opIn) {
          case 1:
            d_reduceCartField<512,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<512,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<512,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 256:
        switch (opIn) {
          case 1:
            d_reduceCartField<256,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<256,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<256,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 128:
        switch (opIn) {
          case 1:
            d_reduceCartField<128,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<128,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<128,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 64:
        switch (opIn) {
          case 1:
            d_reduceCartField<64,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<64,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<64,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 32:
        switch (opIn) {
          case 1:
            d_reduceCartField<32,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<32,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<32,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 16:
        switch (opIn) {
          case 1:
            d_reduceCartField<16,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<16,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<16,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 8:
        switch (opIn) {
          case 1:
            d_reduceCartField<8,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<8,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<8,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 4:
        switch (opIn) {
          case 1:
            d_reduceCartField<4,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<4,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<4,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 2:
        switch (opIn) {
          case 1:
            d_reduceCartField<2,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<2,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<2,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 1:
        switch (opIn) {
          case 1:
            d_reduceCartField<1,true,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<1,true,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<1,true,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
    }
  } else {
    switch (threads) {
      case 512:
        switch (opIn) {
          case 1:
            d_reduceCartField<512,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<512,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<512,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 256:
        switch (opIn) {
          case 1:
            d_reduceCartField<256,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<256,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<256,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 128:
        switch (opIn) {
          case 1:
            d_reduceCartField<128,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<128,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<128,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 64:
        switch (opIn) {
          case 1:
            d_reduceCartField<64,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<64,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<64,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 32:
        switch (opIn) {
          case 1:
            d_reduceCartField<32,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<32,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<32,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 16:
        switch (opIn) {
          case 1:
            d_reduceCartField<16,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<16,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<16,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 8:
        switch (opIn) {
          case 1:
            d_reduceCartField<8,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<8,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<8,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 4:
        switch (opIn) {
          case 1:
            d_reduceCartField<4,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<4,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<4,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 2:
        switch (opIn) {
          case 1:
            d_reduceCartField<2,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<2,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<2,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
      case 1:
        switch (opIn) {
          case 1:
            d_reduceCartField<1,false,binOpMin><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 2:
            d_reduceCartField<1,false,binOpMax><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
          case 3:
            d_reduceCartField<1,false,binOpSum><<<blocks, threads, smemSize>>>(fIn, blockRed);
            break;
        }
        break;
    }
  }
}

void reduceDeviceArray(int opIn, int numElements, int blocks, int threads, double *d_dataIn, double *d_dataOut) {
  // Launch the device kernel that reduces a device array 'd_dataIn'
  // containing 'numElements' elements.

  // When there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

  if (isPow2(numElements)) {
    switch (threads) {
      case 512:
        switch (opIn) {
          case 1:
            d_reduce<512,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<512,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<512,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 256:
        switch (opIn) {
          case 1:
            d_reduce<256,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<256,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<256,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 128:
        switch (opIn) {
          case 1:
            d_reduce<128,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<128,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<128,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 64:
        switch (opIn) {
          case 1:
            d_reduce<64,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<64,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<64,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 32:
        switch (opIn) {
          case 1:
            d_reduce<32,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<32,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<32,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 16:
        switch (opIn) {
          case 1:
            d_reduce<16,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<16,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<16,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 8:
        switch (opIn) {
          case 1:
            d_reduce<8,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<8,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<8,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 4:
        switch (opIn) {
          case 1:
            d_reduce<4,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<4,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<4,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 2:
        switch (opIn) {
          case 1:
            d_reduce<2,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<2,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<2,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 1:
        switch (opIn) {
          case 1:
            d_reduce<1,true,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<1,true,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<1,true,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
    }
  } else {
    switch (threads) {
      case 512:
        switch (opIn) {
          case 1:
            d_reduce<512,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<512,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<512,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 256:
        switch (opIn) {
          case 1:
            d_reduce<256,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<256,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<256,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 128:
        switch (opIn) {
          case 1:
            d_reduce<128,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<128,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<128,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 64:
        switch (opIn) {
          case 1:
            d_reduce<64,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<64,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<64,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 32:
        switch (opIn) {
          case 1:
            d_reduce<32,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<32,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<32,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 16:
        switch (opIn) {
          case 1:
            d_reduce<16,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<16,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<16,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 8:
        switch (opIn) {
          case 1:
            d_reduce<8,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<8,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<8,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 4:
        switch (opIn) {
          case 1:
            d_reduce<4,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<4,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<4,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 2:
        switch (opIn) {
          case 1:
            d_reduce<2,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<2,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<2,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
      case 1:
        switch (opIn) {
          case 1:
            d_reduce<1,false,binOpMin><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 2:
            d_reduce<1,false,binOpMax><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
          case 3:
            d_reduce<1,false,binOpSum><<<blocks,threads,smemSize>>>(d_dataIn, d_dataOut, numElements);
            break;
        }
        break;
    }
  }
}

void cuda_cartFieldReduce(const int reduceOp, int numCellsTot, int numBlocks, int numThreads, int maxBlocks, int maxThreads,
                  GkDeviceProp *prop, GkylCartField_t *fIn, double *blockOut, double *intermediate, double *out) {
  // Find the maximum in the CartField 'fIn' (type double) and place
  // it in the device-memory variable 'out'.
  // This function follows 'reduce6' (using Cooperative Groups) in cuda-samples:
  //   https://github.com/NVIDIA/cuda-samples/tree/master/Samples/reduction
  // The algorithm uses two other temporary variables: 'blockOut' and 
  // 'intermediate' have size=numBlocks and were allocated already.


  // Call the kernel that reduces a CartField (fIn) into a device array (blockOut)
  // which contains the reduction performed by each block.
  reduceCartField(reduceOp, numCellsTot, numBlocks, numThreads, fIn, blockOut);

  // Reduce partial block reductions on GPU.
  int newNum = numBlocks;
  while (newNum > 1) {
    int threads = 0, blocks = 0;

    getNumBlocksAndThreads(prop, newNum, maxBlocks, maxThreads, blocks, threads);

    checkCudaErrors(hipMemcpy(intermediate, blockOut, newNum * sizeof(double), hipMemcpyDeviceToDevice));

    reduceDeviceArray(reduceOp, newNum, blocks, threads, intermediate, blockOut);

    newNum = (newNum + (threads*2-1))/(threads*2);
  }

  hipDeviceSynchronize();
  // Copy final reduction to output variable.
  checkCudaErrors(hipMemcpy(out, blockOut, sizeof(double), hipMemcpyDeviceToDevice));
}

