#include "hip/hip_runtime.h"
/* -*- c++ -*- */

#include <cstdio>
#include "RectCartDeviceImpl.h"

extern "C" 
{
    void unit_sumArray(int numBlocks, int numThreads, int n, double a, double *x, double *y);
    void unit_sayHello();
    void unit_showRange(GkylRange_t *range);
    void unit_showGrid(RectCart_t *grid);
}

__global__ void ker_unit_sumArray(int n, double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i<n) y[i] = a*x[i]+y[i];
}

__global__ void ker_unit_sayHello()
{
  printf("Hello!\n");
}

__global__ void ker_unit_showRange(GkylRange_t *range)
{
  printf("Range ndim: %d\n", range->ndim);
  for (unsigned i=0; i<range->ndim; ++i)
    printf(" %d, %d\n", range->lower[i], range->upper[i]);
}

__global__ void ker_unit_showGrid(RectCart_t *grid)
{
  printf("Grid ndim: %d\n", grid->ndim);
  for (unsigned i=0; i<grid->ndim; ++i)
    printf(" %g, %g\n", grid->lower[i], grid->upper[i]);

  double xc[6];
  int idx[6];
  cellCenter(grid, idx, xc);
}

void unit_sumArray(int numBlocks, int numThreads, int n, double a, double *x, double *y)
{
   ker_unit_sumArray<<<numBlocks, numThreads>>>(n, a, x, y);
}

void unit_sayHello()
{
  ker_unit_sayHello<<<1, 1>>>();
}

void unit_showRange(GkylRange_t *devRange)
{
  ker_unit_showRange<<<1, 1>>>(devRange);
}

void unit_showGrid(RectCart_t *devGrid)
{
  ker_unit_showGrid<<<1, 1>>>(devGrid);
}
