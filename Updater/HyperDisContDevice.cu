#include "hip/hip_runtime.h"
#include <cstdio>
#include <GkylHyperDisCont.h>
#include <GkylVlasov.h>
#include <VlasovModDecl.h>

__global__ void cuda_HyperDisCont(GkylHyperDisCont_t *hyper, GkylCartField_t *fIn, GkylCartField_t *fRhsOut) {

  GkylRange_t *localRange = fIn->localRange;
  unsigned int ndim = localRange->ndim;
  unsigned int numComponents = fRhsOut->numComponents;
  
  // set up indexers for localRange and fIn (localExtRange)
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer fIdxr = fIn->genIndexer();

  // get setup data from GkylHyperDisCont_t structure
  GkylRectCart_t *grid = fIn->grid;
  int *updateDirs = hyper->updateDirs;
  int numUpdateDirs = hyper->numUpdateDirs;
  bool *zeroFluxFlags = hyper->zeroFluxFlags;
  bool clearOut = hyper->clearOut;
  Gkyl::Vlasov *eq = hyper->equation;
  
  // CUDA thread "loop" over (non-ghost) cells in local range
  for(unsigned int linearIdx = threadIdx.x + blockIdx.x*blockDim.x; linearIdx < localRange->volume(); linearIdx += blockDim.x*gridDim.x) {
    int idxC[6];
    int idxL[6];
    int idxR[6];
    
    double xcC[6];
    double xcL[6];
    double xcR[6];

    // get i,j,k... index idxC from linear index linearIdx using localRange invIndexer
    localIdxr.invIndex(linearIdx, idxC);
    // convert i,j,k... index idxC into a linear index linearIdxC
    // note that linearIdxC != linearIdx.
    // this is because linearIdxC will have jumps because of ghost cells
    int linearIdxC = fIdxr.index(idxC);

    grid->cellCenter(idxC, xcC);
    double *dx = grid->dx;
    
    double *fInC = fIn->getDataPtrAt(linearIdxC);
    double *fRhsOutC = fRhsOut->getDataPtrAt(linearIdxC);
    if(clearOut) {
      memset(fRhsOutC, 0., sizeof(double)*numComponents);
    }
    double cflRate = eq->volTerm(xcC, dx, idxC, fInC, fRhsOutC);

    // hard code this size for now. 
    // should be numComponents, but want to avoid dynamic memory alloc
    double dummy[200];
 
    for(int i=0; i<numUpdateDirs; i++) {
      int dir = updateDirs[i] - 1;

      for(int d=0; d<ndim; d++) {
        if(d!=dir) {
          idxL[d] = idxC[d];
          idxR[d] = idxC[d];
        } else {
          idxL[d] = idxC[d] - 1;
          idxR[d] = idxC[d] + 1;
        }
      }

      int linearIdxL = fIdxr.index(idxL);
      int linearIdxR = fIdxr.index(idxR);
      grid->cellCenter(idxL, xcL);
      grid->cellCenter(idxR, xcR);
      double *fInL = fIn->getDataPtrAt(linearIdxL);
      double *fInR = fIn->getDataPtrAt(linearIdxR);
      
      // left (of C) surface update. use dummy in place of fRhsOutL (cell to left of surface) so that only current cell (C) is updated.
      if(!(zeroFluxFlags[dir] && idxC[dir] == localRange->lower[dir])) {
        eq->surfTerm(dir, dummy, dummy, xcL, xcC, dx, dx, 0., idxL, idxC, fInL, fInC, dummy, fRhsOutC);
      } else if( zeroFluxFlags[dir]) {
        eq->boundarySurfTerm(dir, dummy, dummy, xcL, xcC, dx, dx, 0., idxL, idxC, fInL, fInC, dummy, fRhsOutC);
      }

      // right (of C) surface update. use dummy in place of fRhsOutR (cell to left of surface) so that only current cell (C) is updated.
      if(!(zeroFluxFlags[dir] && idxC[dir] == localRange->upper[dir])) {
        eq->surfTerm(dir, dummy, dummy, xcC, xcR, dx, dx, 0., idxC, idxR, fInC, fInR, fRhsOutC, dummy);
      } else if( zeroFluxFlags[dir]) {
        eq->boundarySurfTerm(dir, dummy, dummy, xcC, xcR, dx, dx, 0., idxC, idxR, fInC, fInR, fRhsOutC, dummy);
      }
    }
  }
} 

void advanceOnDevice(int numThreads, int numBlocks, GkylHyperDisCont_t *hyper, GkylCartField_t *fIn, GkylCartField_t *fRhsOut) {
  cuda_HyperDisCont<<<numThreads, numBlocks>>>(hyper, fIn, fRhsOut);
}
