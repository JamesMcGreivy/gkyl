#include "hip/hip_runtime.h"
#include <GkylMomentSrc.h>
#include <string.h>
#include <cstdio>

// Makes indexing cleaner
static const int X = 0;
static const int Y = 1;
static const int Z = 2;

static const int RHO = 0;
static const int MX = 1;
static const int MY = 2;
static const int MZ = 3;
static const int ER = 4;

static const int EX = 0;
static const int EY = 1;
static const int EZ = 2;
static const int BX = 3;
static const int BY = 4;
static const int BZ = 5;
static const int PHIE = 6;

#define fidx(n, c) (3 * (n) + (c))
#define eidx(c) (3 * nFluids + (c))

#define sq(x) ((x) * (x))

#define F2(base,i,j) (base)[(j)*(matSize)+(i)]


__global__ static void cuda_gkylMomentSrcTimeCenteredCublasSetPtrs(
   const int matSize,  double *d_lhs, double *d_rhs, double **d_lhs_ptr,
   double **d_rhs_ptr) {
  // numThreads*numBlocks == numRealCells
  const int linearIdx = threadIdx.x + blockIdx.x*blockDim.x;
  d_lhs_ptr[linearIdx] = d_lhs + (sq(matSize))*linearIdx;
  d_rhs_ptr[linearIdx] = d_rhs + (matSize)*linearIdx;
}


GkylMomentSrcDeviceData_t *cuda_gkylMomentSrcInit(
    const int nFluids, const int numBlocks, const int numThreads) {
  const int matSize = 3 * nFluids + 3;
  GkylMomentSrcDeviceData_t *context = new GkylMomentSrcDeviceData_t[1];
  cublascall(hipblasCreate(&(context->handle)));

  const int batchSize = numThreads*numBlocks;
  // device memory for actuall arrays and vectors
  cudacall(hipMalloc(&context->d_lhs, batchSize*sq(matSize)*sizeof(double)));
  cudacall(hipMalloc(&context->d_rhs, batchSize*matSize*sizeof(double)));
  cudacall(hipMalloc(&context->d_info, batchSize*sizeof(int)));
  // device memory for pointers to the actuall arrays and vectors
  cudacall(hipMalloc(&context->d_lhs_ptr, batchSize*sizeof(double*)));
  cudacall(hipMalloc(&context->d_rhs_ptr, batchSize*sizeof(double*)));

  cuda_gkylMomentSrcTimeCenteredCublasSetPtrs<<<numBlocks, numThreads>>>(
      matSize, context->d_lhs, context->d_rhs, context->d_lhs_ptr,
      context->d_rhs_ptr);

  return context;
}


void cuda_gkylMomentSrcDestroy(GkylMomentSrcDeviceData_t *context) {
  cudacall(hipFree(context->d_lhs_ptr));
  cudacall(hipFree(context->d_rhs_ptr));
  cudacall(hipFree(context->d_lhs));
  cudacall(hipFree(context->d_rhs));
  cudacall(hipFree(context->d_info));
  cublascall(hipblasDestroy(context->handle));
}


__device__ static void cuda_gkylMomentSrcTimeCenteredUpdateRhovE(
    const int linearIdx, const int linearIdxC, MomentSrcData_t *sd,
    FluidData_t *fd, double dt, GkylCartField_t **fluidFlds,
    GkylCartField_t *emFld, double **d_rhs_ptr) {
  const double *sol = d_rhs_ptr[linearIdx];
  const int nFluids = sd->nFluids;

  //------------> update solution for fluids
  double chargeDens = 0.0;
  for (int n=0; n<nFluids; ++n)
  {
    double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    double qbym = fd[n].charge/fd[n].mass;

    chargeDens += qbym*f[RHO];

    f[MX] = 2*sol[fidx(n,X)]/qbym - f[MX];
    f[MY] = 2*sol[fidx(n,Y)]/qbym - f[MY];
    f[MZ] = 2*sol[fidx(n,Z)]/qbym - f[MZ];
  }

  //------------> update electric field
  double *em = emFld->getDataPtrAt(linearIdxC);
  em[EX] = 2*sol[eidx(X)] - em[EX];
  em[EY] = 2*sol[eidx(Y)] - em[EY];
  em[EZ] = 2*sol[eidx(Z)] - em[EZ];

  //------------> update correction potential
  const double crhoc = sd->chi_e*chargeDens/sd->epsilon0;
  em[PHIE] += dt*crhoc;
}


__global__ static void cuda_gkylMomentSrcTimeCenteredUpdate(
    MomentSrcData_t *sd, FluidData_t *fd, double dt,
    GkylCartField_t **fluidFlds, GkylCartField_t *emFld, double **d_rhs_ptr) {
  GkylRange_t *localRange = emFld->localRange;
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer fIdxr = emFld->genIndexer();

  // numThreads*numBlocks == numRealCells
  const int linearIdx = threadIdx.x + blockIdx.x*blockDim.x;
  int idxC[3];
  localIdxr.invIndex(linearIdx, idxC);
  const int linearIdxC = fIdxr.index(idxC);
  
  const int nFluids = sd->nFluids;

  double keOld[2]; // XXX
  for (int n = 0; n < nFluids; ++n)
  {
    double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    if (!fd[n].evolve)
      continue;
    keOld[n] = 0.5 * (sq(f[MX]) + sq(f[MY]) + sq(f[MZ])) / f[RHO];
  }

  cuda_gkylMomentSrcTimeCenteredUpdateRhovE(
      linearIdx, linearIdxC, sd, fd, dt, fluidFlds, emFld, d_rhs_ptr);

  if (sd->hasPressure)
  {
    for (int n = 0; n < nFluids; ++n)
    {
      if (!fd[n].evolve)
        continue;
      double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
      const double keNew = 0.5 * (sq(f[MX]) + sq(f[MY]) + sq(f[MZ])) / f[RHO];
      f[ER] += keNew - keOld[n];
    }
  } 
}


__global__ static void cuda_gkylMomentSrcTimeCenteredCublasSetMat(
    MomentSrcData_t *sd, FluidData_t *fd, double dt,
    GkylCartField_t **fluidFlds, GkylCartField_t *emFld, double **d_lhs_ptr,
    double **d_rhs_ptr) {
  GkylRange_t *localRange = emFld->localRange;
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer fIdxr = emFld->genIndexer();

  const int nFluids = sd->nFluids;
  const int matSize = 3 * nFluids + 3;

  // numThreads*numBlocks == numRealCells
  const int linearIdx = threadIdx.x + blockIdx.x*blockDim.x;
  int idxC[3];
  localIdxr.invIndex(linearIdx, idxC);
  const int linearIdxC = fIdxr.index(idxC);
  const double *em = emFld->getDataPtrAt(linearIdxC);

  double *lhs = d_lhs_ptr[linearIdx];
  double *rhs = d_rhs_ptr[linearIdx];

  for (int c=0; c<sq(matSize); c++)
    lhs[c] = 0;

  double dt1 = 0.5 * dt;
  double dt2 = 0.5 * dt / sd->epsilon0;

  for (int n=0; n<nFluids; ++n)
  {
    double qbym = fd[n].charge/fd[n].mass;
    double qbym2 = sq(qbym);

    const double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    if (fd[n].evolve) {
      // off-diagonal elements of lhs
      // eqn. for X-component of current
      F2(lhs, fidx(n,X), fidx(n,Y)) = -dt1*qbym*(em[BZ]);
      F2(lhs, fidx(n,X), fidx(n,Z)) = dt1*qbym*(em[BY]);
      F2(lhs, fidx(n,X), eidx(X)) = -dt1*qbym2*f[RHO];

      // eqn. for Y-component of current
      F2(lhs, fidx(n,Y), fidx(n,X)) = dt1*qbym*(em[BZ]);
      F2(lhs, fidx(n,Y), fidx(n,Z)) = -dt1*qbym*(em[BX]);
      F2(lhs, fidx(n,Y), eidx(Y)) = -dt1*qbym2*f[RHO];

      // eqn. for Z-component of current
      F2(lhs, fidx(n,Z), fidx(n,X)) = -dt1*qbym*(em[BY]);
      F2(lhs, fidx(n,Z), fidx(n,Y)) = dt1*qbym*(em[BX]);
      F2(lhs, fidx(n,Z), eidx(Z)) = -dt1*qbym2*f[RHO];
    }
    // diagonal elements of lhs
    F2(lhs, fidx(n,X), fidx(n,X)) = 1.0;
    F2(lhs, fidx(n,Y), fidx(n,Y)) = 1.0;
    F2(lhs, fidx(n,Z), fidx(n,Z)) = 1.0;

    // fill corresponding RHS elements
    rhs[fidx(n,X)] = qbym*f[MX];
    rhs[fidx(n,Y)] = qbym*f[MY];
    rhs[fidx(n,Z)] = qbym*f[MZ];

    // set current contribution to electric field equation
    F2(lhs, eidx(X), fidx(n,X)) = dt2;
    F2(lhs, eidx(Y), fidx(n,Y)) = dt2;
    F2(lhs, eidx(Z), fidx(n,Z)) = dt2;
  }

  // fill in elements for electric field equations
  F2(lhs, eidx(EX), eidx(EX)) = 1.0;
  F2(lhs, eidx(EY), eidx(EY)) = 1.0;
  F2(lhs, eidx(EZ), eidx(EZ)) = 1.0;

  rhs[eidx(EX)] = em[EX];
  rhs[eidx(EY)] = em[EY];
  rhs[eidx(EZ)] = em[EZ];
}


static void cuda_gkylMomentSrcTimeCenteredCublas(
    int numBlocks, int numThreads, MomentSrcData_t *sd, FluidData_t *fd,
    double dt, GkylCartField_t **fluidFlds, GkylCartField_t *emFld,
    GkylMomentSrcDeviceData_t *context) {
  const int nFluids = sd->nFluids;
  const int matSize = 3 * nFluids + 3;

  double **d_lhs_ptr = context->d_lhs_ptr;
  double **d_rhs_ptr = context->d_rhs_ptr;
  int *d_info = context->d_info;
  hipblasHandle_t &handle = context->handle;

  int batchSize = numThreads*numBlocks;

  cuda_gkylMomentSrcTimeCenteredCublasSetMat<<<numBlocks, numThreads>>>(
      sd, fd, dt, fluidFlds, emFld, d_lhs_ptr, d_rhs_ptr);

  cublascall(hipblasDgetrfBatched(
      handle,
      matSize,  // n
      d_lhs_ptr,  // A
      matSize,  // lda
      NULL,  // int *PivotArray
      d_info,  // int *infoArray
      batchSize // number of pointers contained in A
      ));

  int info;
  cublascall(hipblasDgetrsBatched(
      handle,
      HIPBLAS_OP_N,  // trans
      matSize,  // n
      1,  // nrhs
      d_lhs_ptr,  // matrix A
      matSize,  // lda
      NULL,  // const int *devIpiv
      d_rhs_ptr,  // double *Barray[]
      matSize,  // ldb
      &info,  // int *info
      batchSize // number of pointers contained in A
      ));

  // update solution
  cuda_gkylMomentSrcTimeCenteredUpdate<<<numBlocks, numThreads>>>(
      sd, fd, dt, fluidFlds, emFld, d_rhs_ptr);
}


__global__ static void cuda_gkylMomentSrcTimeCenteredDirect(
    int numBlocks, int numThreads, MomentSrcData_t *sd, FluidData_t *fd,
    double dt, GkylCartField_t **fluidFlds, GkylCartField_t *emFld,
    GkylMomentSrcDeviceData_t *context) {
  GkylRange_t *localRange = emFld->localRange;
  Gkyl::GenIndexer localIdxr(localRange);
  Gkyl::GenIndexer fIdxr = emFld->genIndexer();

  // numThreads*numBlocks == numRealCells
  const int linearIdx = threadIdx.x + blockIdx.x*blockDim.x;
  int idxC[3];
  localIdxr.invIndex(linearIdx, idxC);
  const int linearIdxC = fIdxr.index(idxC);
  double *em = emFld->getDataPtrAt(linearIdxC);

  const int nFluids = sd->nFluids;
  const double epsilon0 = sd->epsilon0;

  double keOld[2]; // XXX
  for (int n = 0; n < nFluids; ++n)
  {
    double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    if (!fd[n].evolve)
      continue;
    keOld[n] = 0.5 * (sq(f[MX]) + sq(f[MY]) + sq(f[MZ])) / f[RHO];
  }

  const double Bx = (em[BX]);
  const double By = (em[BY]);
  const double Bz = (em[BZ]);
  const double Bmag = std::sqrt(Bx*Bx + By*By + Bz*Bz);
  double b[] = {0, 0, 0};
  if (Bmag > 0)
  {
    b[0] = Bx / Bmag;
    b[1] = By / Bmag;
    b[2] = Bz / Bmag;
  }

  extern __shared__ double dummy[];
  int base = 0;

  base += nFluids*blockDim.x;
  double *qbym = dummy + base + threadIdx.x;

  base += nFluids*3*blockDim.x;
  double *JJ = dummy + base + threadIdx.x;

  base += nFluids*blockDim.x;
  double *Wc_dt = dummy + base + threadIdx.x;

  base += nFluids*blockDim.x;
  double *wp_dt2 = dummy + base + threadIdx.x;

  double K[] = {0, 0, 0};
  double w02 = 0.;
  double gam2 = 0.;
  double delta = 0.;

  for (int n=0; n < nFluids; ++n)
  {
    qbym[n] = fd[n].charge / fd[n].mass;
    const double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    double *J = JJ+n*3;
    J[0] = f[MX] * qbym[n];
    J[1] = f[MY] * qbym[n];
    J[2] = f[MZ] * qbym[n];
    if (!fd[n].evolve)
      continue;
    Wc_dt[n] = qbym[n] * Bmag * dt;
    wp_dt2[n] = f[RHO] * sq(qbym[n]) / epsilon0 * sq(dt);
    double tmp = 1. + sq(Wc_dt[n]) / 4.;
    w02 += wp_dt2[n] / tmp;
    gam2 += wp_dt2[n] * sq(Wc_dt[n]) / tmp;
    delta += wp_dt2[n] * Wc_dt[n] / tmp;

    double bDotJ = b[0]*J[0] + b[1]*J[1] + b[2]*J[2];
    double bCrossJ[] = {
      b[1]*J[2]-b[2]*J[1], // by*Jz-bz*Jy
      b[2]*J[0]-b[0]*J[2], // bz*Jx-bx*Jz
      b[0]*J[1]-b[1]*J[0], // bx*Jy-by*Jx
    };

#pragma unroll
    for(int c=0; c<3; c++) {
      K[c] -= dt / tmp * (J[c] + sq(Wc_dt[c] / 2.) * b[c] * bDotJ
              - (Wc_dt[n] / 2.) * bCrossJ[c]);
    }
  }
  double Delta2 = sq(delta) / (1. + w02 / 4.);

  const double F[] = {em[EX] * epsilon0, em[EY] * epsilon0, em[EZ] * epsilon0};
  double F_halfK[3];
#pragma unroll
  for (int c=0; c<3; c++) {
    F_halfK[c] = F[c] + 0.5 * K[c];
    for (int n=0; n < nFluids; ++n)
    {
      if (fd[n].evolve)
        continue;
      F_halfK[c] -= (0.5 * dt) * JJ[n*3+c];
    }
  }

  const double tmp = 1. / (1. + w02 / 4. + Delta2 / 64.);
  double bDotF_halfK = b[0]*F_halfK[0] + b[1]*F_halfK[1] + b[2]*F_halfK[2];
  double bCrossF_halfK[] = {
    b[1]*F_halfK[2]-b[2]*F_halfK[1], // by*Fz-bz*Fy
    b[2]*F_halfK[0]-b[0]*F_halfK[2], // bz*Fx-bx*Fz
    b[0]*F_halfK[1]-b[1]*F_halfK[0], // bx*Fy-by*Fx
  };

  double Fbar[3];
#pragma unroll
  for (int c=0; c<3; c++) {
    Fbar[c] = tmp * (
      F_halfK[c]
      + ((Delta2 / 64. - gam2 / 16.) / (1. + w02 / 4. + gam2 / 16.))
         * b[c] * bDotF_halfK
      + (delta / 8. / (1. + w02 / 4.)) * bCrossF_halfK[c]
      );
  } 

  double F_new[3];
#pragma unroll
  for (int c=0; c<3; c++) {
    F_new[c] = 2. * Fbar[c] - F[c];
  }
  em[EX] = F_new[0] / epsilon0;
  em[EY] = F_new[1] / epsilon0;
  em[EZ] = F_new[2] / epsilon0;

  double chargeDens = 0.0;
  for (int n = 0; n < nFluids; ++n)
  {
    double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
    chargeDens += qbym[n] * f[RHO];
    if (!fd[n].evolve)
      continue;

    double *J = JJ+n*3;
    double Jstar[3];
    double J_new[3];

#pragma unroll
    for (int c=0; c<3; c++) {
      Jstar[c] = J[c] + Fbar[c] * (wp_dt2[n] / dt / 2.);
    }
    double bDotJstar = b[0]*Jstar[0] + b[1]*Jstar[1] + b[2]*Jstar[2];
    double bCrossJstar[] = {
      b[1]*Jstar[2]-b[2]*Jstar[1], // by*Jz-bz*Jy
      b[2]*Jstar[0]-b[0]*Jstar[2], // bz*Jx-bx*Jz
      b[0]*Jstar[1]-b[1]*Jstar[0], // bx*Jy-by*Jx
    };

#pragma unroll
    for (int c=0; c<3; c++) {
      J_new[c] = 2. * (Jstar[c] + sq(Wc_dt[n] / 2.) * b[c] * bDotJstar
                 - (Wc_dt[n] / 2.) * bCrossJstar[c]) / (1. + sq(Wc_dt[n] / 2.))
                 - J[c];
    }

    f[MX] = J_new[0] / qbym[n];
    f[MY] = J_new[1] / qbym[n];
    f[MZ] = J_new[2] / qbym[n];
  } 

  double crhoc = sd->chi_e * chargeDens/sd->epsilon0;
  em[PHIE] += dt * crhoc;

  if (sd->hasPressure)
  {
    for (int n = 0; n < nFluids; ++n)
    {
      if (!fd[n].evolve)
        continue;
      double *f = fluidFlds[n]->getDataPtrAt(linearIdxC);
      const double keNew = 0.5 * (sq(f[MX]) + sq(f[MY]) + sq(f[MZ])) / f[RHO];
      f[ER] += keNew - keOld[n];
    }
  } 
}


void momentSrcAdvanceOnDevice(
    const int nFluids, const int numBlocks, const int numThreads,
    MomentSrcData_t *sd, FluidData_t *fd, double dt,
    GkylCartField_t **fluidFlds, GkylCartField_t *emFld, const char *scheme,
    GkylMomentSrcDeviceData_t *context)
{
  if (strcmp(scheme, "time-centered")==0) {
    cuda_gkylMomentSrcTimeCenteredCublas(
        numBlocks, numThreads, sd, fd, dt, fluidFlds, emFld, context);
  } else if (strcmp(scheme, "time-centered-direct")==0
             || strcmp(scheme, "direct")==0) {
    int sharedMemSize = 0;
    // qbym, J, Wc_dt, wp_dt2
    sharedMemSize += numThreads * nFluids * (1 + 3 + 1 + 1);
    sharedMemSize *= sizeof(double);

    cuda_gkylMomentSrcTimeCenteredDirect
      <<<numBlocks, numThreads, sharedMemSize>>>(
      numBlocks, numThreads, sd, fd, dt, fluidFlds, emFld, context);
  }
}

