#include "hip/hip_runtime.h"
/* -*- c++ -*- */
// Gkyl ------------------------------------------------------------------------
//
// CUDA wrappers for moment-calculating kernels.
//
//    _______     ___
// + 6 @ |||| # P ||| +
//------------------------------------------------------------------------------

#include "DistFuncMomentCalcDeviceWrappers.h"
#include "DistFuncMomentCalcModDecl.h"
#include "DistFuncMomentCalcDeviceCommon.cu"


 __global__ void d_calcMom1x1vSer_M0_P1(GkylCartField_t *fIn, GkylCartField_t *out) {
  // In computing moments we will first assign whole configuration-space cells to a single block.
  // Then one must perform a reduction across a block for each conf-space basis coefficient.

  // Index of the first phase-space memory address to access.
  unsigned int linearIdx       = blockIdx.x*blockDim.x + threadIdx.x;
  GkylRange_t *localPhaseRange = fIn->localRange;
  Gkyl::GenIndexer localPhaseIdxr(localPhaseRange);
  Gkyl::GenIndexer fIdxr = fIn->genIndexer();

  int phaseIdx[2];
  localPhaseIdxr.invIndex(linearIdx, phaseIdx);
  int phaseLinIdx = fIdxr.index(phaseIdx);

  double localSum[2];
  for (unsigned int k = 0; k < 2; k++) {
    localSum[k]=0.0;
  }

  // Pointers to quantities expected by the moment kernel.
  const double *distF  = fIn->getDataPtrAt(phaseLinIdx);
  GkylRectCart_t *grid = fIn->grid;
  double cellxc[2];
  grid->cellCenter(phaseIdx,cellxc);
  double *cellCenter  = &cellxc[0];
  double *cellSize    = &grid->dx[0];
  double *localSumPtr = &localSum[0];

  MomentCalc1x1vSer_M0_P1(cellCenter, cellSize, distF, localSumPtr);

  blockReduceComponentsSum(localSumPtr, 2);

  // Configuration space indexes.
  GkylRange_t *localConfRange = out->localRange;
  Gkyl::GenIndexer localConfIdxr(localConfRange);
  Gkyl::GenIndexer outIdxr = out->genIndexer();
  int confIdx[1];
  for (unsigned int k = 0; k < 1; k++) {
    confIdx[k] = phaseIdx[k];
  }
  int confLinIdx = outIdxr.index(phaseIdx);
  double *mom    = out->getDataPtrAt(confLinIdx);
  unsigned int numComponents = out->numComponents;

  if (threadIdx.x==0) {
    for (unsigned int k = 0; k < numComponents; k++) {
      mom[confLinIdx+k] = localSumPtr[k];
    }
  }

}


 void cuda_MomentCalc1x1vSer_M0_P1(GkDeviceProp *prop, int numBlocks, int numThreads, GkylCartField_t *fIn, GkylCartField_t *out) {
  int warpSize = prop->warpSize;

  d_calcMom1x1vSer_M0_P1<<<numBlocks, numThreads, 2*(numThreads/warpSize)*sizeof(double)>>>(fIn, out);
}


 __global__ void d_calcMom1x1vSer_M0_P2(GkylCartField_t *fIn, GkylCartField_t *out) {
  // In computing moments we will first assign whole configuration-space cells to a single block.
  // Then one must perform a reduction across a block for each conf-space basis coefficient.

  // Index of the first phase-space memory address to access.
  unsigned int linearIdx       = blockIdx.x*blockDim.x + threadIdx.x;
  GkylRange_t *localPhaseRange = fIn->localRange;
  Gkyl::GenIndexer localPhaseIdxr(localPhaseRange);
  Gkyl::GenIndexer fIdxr = fIn->genIndexer();

  int phaseIdx[2];
  localPhaseIdxr.invIndex(linearIdx, phaseIdx);
  int phaseLinIdx = fIdxr.index(phaseIdx);

  double localSum[3];
  for (unsigned int k = 0; k < 3; k++) {
    localSum[k]=0.0;
  }

  // Pointers to quantities expected by the moment kernel.
  const double *distF  = fIn->getDataPtrAt(phaseLinIdx);
  GkylRectCart_t *grid = fIn->grid;
  double cellxc[2];
  grid->cellCenter(phaseIdx,cellxc);
  double *cellCenter  = &cellxc[0];
  double *cellSize    = &grid->dx[0];
  double *localSumPtr = &localSum[0];

  MomentCalc1x1vSer_M0_P2(cellCenter, cellSize, distF, localSumPtr);

  blockReduceComponentsSum(localSumPtr, 3);

  // Configuration space indexes.
  GkylRange_t *localConfRange = out->localRange;
  Gkyl::GenIndexer localConfIdxr(localConfRange);
  Gkyl::GenIndexer outIdxr = out->genIndexer();
  int confIdx[1];
  for (unsigned int k = 0; k < 1; k++) {
    confIdx[k] = phaseIdx[k];
  }
  int confLinIdx = outIdxr.index(phaseIdx);
  double *mom    = out->getDataPtrAt(confLinIdx);
  unsigned int numComponents = out->numComponents;

  if (threadIdx.x==0) {
    for (unsigned int k = 0; k < numComponents; k++) {
      mom[confLinIdx+k] = localSumPtr[k];
    }
  }

}


 void cuda_MomentCalc1x1vSer_M0_P2(GkDeviceProp *prop, int numBlocks, int numThreads, GkylCartField_t *fIn, GkylCartField_t *out) {
  int warpSize = prop->warpSize;

  d_calcMom1x1vSer_M0_P2<<<numBlocks, numThreads, 3*(numThreads/warpSize)*sizeof(double)>>>(fIn, out);
}
