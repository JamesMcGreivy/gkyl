
#include <hip/hip_runtime.h>
//#include <VlasovModDecl.h> 
__host__ __device__ double VlasovSurfElcMag2x3vSer_VX_P1(const double* __restrict__ wl, const double* __restrict__ wr, const double* __restrict__ dxvl, const double* __restrict__ dxvr, const double amax, const double* __restrict__ EM, const double* __restrict__ fl, const double* __restrict__ fr, double *outl, double *outr) 
{ 
// w: Cell-center coordinates. dxv[NDIM]: Cell spacing. amax: amax in global lax flux. E: EM field. fl/fr: Distribution function in left/right cells 
// outl/outr: output distribution function in left/right cells 
// returns abs(amid) for use in determining amax in cfl and global lax flux 
  double dv10l = 2/dxvl[2]; 
  double dv10r = 2/dxvr[2]; 
  const double *E0 = &EM[0]; 
  const double dv1 = dxvr[2], wv1 = wr[2]; 
  const double dv2 = dxvr[3], wv2 = wr[3]; 
  const double dv3 = dxvr[4], wv3 = wr[4]; 
  const double *B0 = &EM[12]; 
  const double *B1 = &EM[16]; 
  const double *B2 = &EM[20]; 

  double Ghat[16]; 
  double favg[16]; 
  double alpha[16]; 

  favg[0] = (-1.224744871391589*fr[3])+1.224744871391589*fl[3]+0.7071067811865475*fr[0]+0.7071067811865475*fl[0]; 
  favg[1] = (-1.224744871391589*fr[7])+1.224744871391589*fl[7]+0.7071067811865475*fr[1]+0.7071067811865475*fl[1]; 
  favg[2] = (-1.224744871391589*fr[8])+1.224744871391589*fl[8]+0.7071067811865475*fr[2]+0.7071067811865475*fl[2]; 
  favg[3] = (-1.224744871391589*fr[11])+1.224744871391589*fl[11]+0.7071067811865475*fr[4]+0.7071067811865475*fl[4]; 
  favg[4] = (-1.224744871391589*fr[14])+1.224744871391589*fl[14]+0.7071067811865475*fr[5]+0.7071067811865475*fl[5]; 
  favg[5] = (-1.224744871391589*fr[16])+1.224744871391589*fl[16]+0.7071067811865475*fr[6]+0.7071067811865475*fl[6]; 
  favg[6] = (-1.224744871391589*fr[18])+1.224744871391589*fl[18]+0.7071067811865475*fr[9]+0.7071067811865475*fl[9]; 
  favg[7] = (-1.224744871391589*fr[19])+1.224744871391589*fl[19]+0.7071067811865475*fr[10]+0.7071067811865475*fl[10]; 
  favg[8] = (-1.224744871391589*fr[21])+1.224744871391589*fl[21]+0.7071067811865475*fr[12]+0.7071067811865475*fl[12]; 
  favg[9] = (-1.224744871391589*fr[22])+1.224744871391589*fl[22]+0.7071067811865475*fr[13]+0.7071067811865475*fl[13]; 
  favg[10] = (-1.224744871391589*fr[25])+1.224744871391589*fl[25]+0.7071067811865475*fr[15]+0.7071067811865475*fl[15]; 
  favg[11] = (-1.224744871391589*fr[26])+1.224744871391589*fl[26]+0.7071067811865475*fr[17]+0.7071067811865475*fl[17]; 
  favg[12] = (-1.224744871391589*fr[27])+1.224744871391589*fl[27]+0.7071067811865475*fr[20]+0.7071067811865475*fl[20]; 
  favg[13] = (-1.224744871391589*fr[29])+1.224744871391589*fl[29]+0.7071067811865475*fr[23]+0.7071067811865475*fl[23]; 
  favg[14] = (-1.224744871391589*fr[30])+1.224744871391589*fl[30]+0.7071067811865475*fr[24]+0.7071067811865475*fl[24]; 
  favg[15] = (-1.224744871391589*fr[31])+1.224744871391589*fl[31]+0.7071067811865475*fr[28]+0.7071067811865475*fl[28]; 

  alpha[0] = 2.0*(B2[0]*wv2+E0[0])-2.0*B1[0]*wv3; 
  alpha[1] = 2.0*(B2[1]*wv2+E0[1])-2.0*B1[1]*wv3; 
  alpha[2] = 2.0*(B2[2]*wv2+E0[2])-2.0*B1[2]*wv3; 
  alpha[3] = 0.5773502691896258*B2[0]*dv2; 
  alpha[4] = -0.5773502691896258*B1[0]*dv3; 
  alpha[5] = 2.0*(B2[3]*wv2+E0[3])-2.0*B1[3]*wv3; 
  alpha[6] = 0.5773502691896258*B2[1]*dv2; 
  alpha[7] = 0.5773502691896258*B2[2]*dv2; 
  alpha[8] = -0.5773502691896258*B1[1]*dv3; 
  alpha[9] = -0.5773502691896258*B1[2]*dv3; 
  alpha[11] = 0.5773502691896258*B2[3]*dv2; 
  alpha[12] = -0.5773502691896258*B1[3]*dv3; 

  const double amid = 0.25*alpha[0]; 

  Ghat[0] = 0.3535533905932737*(1.732050807568877*(fr[3]+fl[3])-1.0*fr[0]+fl[0])*amax+0.125*(alpha[12]*favg[12]+alpha[11]*favg[11]+alpha[9]*favg[9]+alpha[8]*favg[8]+alpha[7]*favg[7]+alpha[6]*favg[6]+alpha[5]*favg[5]+alpha[4]*favg[4]+alpha[3]*favg[3]+alpha[2]*favg[2]+alpha[1]*favg[1]+alpha[0]*favg[0]); 
  Ghat[1] = 0.3535533905932737*(1.732050807568877*(fr[7]+fl[7])-1.0*fr[1]+fl[1])*amax+0.125*(alpha[9]*favg[12]+favg[9]*alpha[12]+alpha[7]*favg[11]+favg[7]*alpha[11]+alpha[4]*favg[8]+favg[4]*alpha[8]+alpha[3]*favg[6]+favg[3]*alpha[6]+alpha[2]*favg[5]+favg[2]*alpha[5]+alpha[0]*favg[1]+favg[0]*alpha[1]); 
  Ghat[2] = 0.3535533905932737*(1.732050807568877*(fr[8]+fl[8])-1.0*fr[2]+fl[2])*amax+0.125*(alpha[8]*favg[12]+favg[8]*alpha[12]+alpha[6]*favg[11]+favg[6]*alpha[11]+alpha[4]*favg[9]+favg[4]*alpha[9]+alpha[3]*favg[7]+favg[3]*alpha[7]+alpha[1]*favg[5]+favg[1]*alpha[5]+alpha[0]*favg[2]+favg[0]*alpha[2]); 
  Ghat[3] = 0.3535533905932737*(1.732050807568877*(fr[11]+fl[11])-1.0*fr[4]+fl[4])*amax+0.125*(alpha[12]*favg[15]+alpha[9]*favg[14]+alpha[8]*favg[13]+alpha[5]*favg[11]+favg[5]*alpha[11]+alpha[4]*favg[10]+alpha[2]*favg[7]+favg[2]*alpha[7]+alpha[1]*favg[6]+favg[1]*alpha[6]+alpha[0]*favg[3]+favg[0]*alpha[3]); 
  Ghat[4] = 0.3535533905932737*(1.732050807568877*(fr[14]+fl[14])-1.0*fr[5]+fl[5])*amax+0.125*(alpha[11]*favg[15]+alpha[7]*favg[14]+alpha[6]*favg[13]+alpha[5]*favg[12]+favg[5]*alpha[12]+alpha[3]*favg[10]+alpha[2]*favg[9]+favg[2]*alpha[9]+alpha[1]*favg[8]+favg[1]*alpha[8]+alpha[0]*favg[4]+favg[0]*alpha[4]); 
  Ghat[5] = 0.3535533905932737*(1.732050807568877*(fr[16]+fl[16])-1.0*fr[6]+fl[6])*amax+0.125*(alpha[4]*favg[12]+favg[4]*alpha[12]+alpha[3]*favg[11]+favg[3]*alpha[11]+alpha[8]*favg[9]+favg[8]*alpha[9]+alpha[6]*favg[7]+favg[6]*alpha[7]+alpha[0]*favg[5]+favg[0]*alpha[5]+alpha[1]*favg[2]+favg[1]*alpha[2]); 
  Ghat[6] = 0.3535533905932737*(1.732050807568877*(fr[18]+fl[18])-1.0*fr[9]+fl[9])*amax+0.125*(alpha[9]*favg[15]+alpha[12]*favg[14]+alpha[4]*favg[13]+alpha[2]*favg[11]+favg[2]*alpha[11]+alpha[8]*favg[10]+alpha[5]*favg[7]+favg[5]*alpha[7]+alpha[0]*favg[6]+favg[0]*alpha[6]+alpha[1]*favg[3]+favg[1]*alpha[3]); 
  Ghat[7] = 0.3535533905932737*(1.732050807568877*(fr[19]+fl[19])-1.0*fr[10]+fl[10])*amax+0.125*(alpha[8]*favg[15]+alpha[4]*favg[14]+alpha[12]*favg[13]+alpha[1]*favg[11]+favg[1]*alpha[11]+alpha[9]*favg[10]+alpha[0]*favg[7]+favg[0]*alpha[7]+alpha[5]*favg[6]+favg[5]*alpha[6]+alpha[2]*favg[3]+favg[2]*alpha[3]); 
  Ghat[8] = 0.3535533905932737*(1.732050807568877*(fr[21]+fl[21])-1.0*fr[12]+fl[12])*amax+0.125*(alpha[7]*favg[15]+alpha[11]*favg[14]+alpha[3]*favg[13]+alpha[2]*favg[12]+favg[2]*alpha[12]+alpha[6]*favg[10]+alpha[5]*favg[9]+favg[5]*alpha[9]+alpha[0]*favg[8]+favg[0]*alpha[8]+alpha[1]*favg[4]+favg[1]*alpha[4]); 
  Ghat[9] = 0.3535533905932737*(1.732050807568877*(fr[22]+fl[22])-1.0*fr[13]+fl[13])*amax+0.125*(alpha[6]*favg[15]+alpha[3]*favg[14]+alpha[11]*favg[13]+alpha[1]*favg[12]+favg[1]*alpha[12]+alpha[7]*favg[10]+alpha[0]*favg[9]+favg[0]*alpha[9]+alpha[5]*favg[8]+favg[5]*alpha[8]+alpha[2]*favg[4]+favg[2]*alpha[4]); 
  Ghat[10] = 0.3535533905932737*(1.732050807568877*(fr[25]+fl[25])-1.0*fr[15]+fl[15])*amax+0.125*(alpha[5]*favg[15]+alpha[2]*favg[14]+alpha[1]*favg[13]+alpha[11]*favg[12]+favg[11]*alpha[12]+alpha[0]*favg[10]+alpha[7]*favg[9]+favg[7]*alpha[9]+alpha[6]*favg[8]+favg[6]*alpha[8]+alpha[3]*favg[4]+favg[3]*alpha[4]); 
  Ghat[11] = 0.3535533905932737*(1.732050807568877*(fr[26]+fl[26])-1.0*fr[17]+fl[17])*amax+0.125*(alpha[4]*favg[15]+alpha[8]*favg[14]+alpha[9]*favg[13]+favg[10]*alpha[12]+alpha[0]*favg[11]+favg[0]*alpha[11]+alpha[1]*favg[7]+favg[1]*alpha[7]+alpha[2]*favg[6]+favg[2]*alpha[6]+alpha[3]*favg[5]+favg[3]*alpha[5]); 
  Ghat[12] = 0.3535533905932737*(1.732050807568877*(fr[27]+fl[27])-1.0*fr[20]+fl[20])*amax+0.125*(alpha[3]*favg[15]+alpha[6]*favg[14]+alpha[7]*favg[13]+alpha[0]*favg[12]+favg[0]*alpha[12]+favg[10]*alpha[11]+alpha[1]*favg[9]+favg[1]*alpha[9]+alpha[2]*favg[8]+favg[2]*alpha[8]+alpha[4]*favg[5]+favg[4]*alpha[5]); 
  Ghat[13] = 0.3535533905932737*(1.732050807568877*(fr[29]+fl[29])-1.0*fr[23]+fl[23])*amax+0.125*(alpha[2]*favg[15]+alpha[5]*favg[14]+alpha[0]*favg[13]+alpha[7]*favg[12]+favg[7]*alpha[12]+alpha[9]*favg[11]+favg[9]*alpha[11]+alpha[1]*favg[10]+alpha[3]*favg[8]+favg[3]*alpha[8]+alpha[4]*favg[6]+favg[4]*alpha[6]); 
  Ghat[14] = 0.3535533905932737*(1.732050807568877*(fr[30]+fl[30])-1.0*fr[24]+fl[24])*amax+0.125*(alpha[1]*favg[15]+alpha[0]*favg[14]+alpha[5]*favg[13]+alpha[6]*favg[12]+favg[6]*alpha[12]+alpha[8]*favg[11]+favg[8]*alpha[11]+alpha[2]*favg[10]+alpha[3]*favg[9]+favg[3]*alpha[9]+alpha[4]*favg[7]+favg[4]*alpha[7]); 
  Ghat[15] = 0.3535533905932737*(1.732050807568877*(fr[31]+fl[31])-1.0*fr[28]+fl[28])*amax+0.125*(alpha[0]*favg[15]+alpha[1]*favg[14]+alpha[2]*favg[13]+alpha[3]*favg[12]+favg[3]*alpha[12]+alpha[4]*favg[11]+favg[4]*alpha[11]+alpha[5]*favg[10]+alpha[6]*favg[9]+favg[6]*alpha[9]+alpha[7]*favg[8]+favg[7]*alpha[8]); 

  outr[0] += 0.7071067811865475*Ghat[0]*dv10r; 
  outr[1] += 0.7071067811865475*Ghat[1]*dv10r; 
  outr[2] += 0.7071067811865475*Ghat[2]*dv10r; 
  outr[3] += -1.224744871391589*Ghat[0]*dv10r; 
  outr[4] += 0.7071067811865475*Ghat[3]*dv10r; 
  outr[5] += 0.7071067811865475*Ghat[4]*dv10r; 
  outr[6] += 0.7071067811865475*Ghat[5]*dv10r; 
  outr[7] += -1.224744871391589*Ghat[1]*dv10r; 
  outr[8] += -1.224744871391589*Ghat[2]*dv10r; 
  outr[9] += 0.7071067811865475*Ghat[6]*dv10r; 
  outr[10] += 0.7071067811865475*Ghat[7]*dv10r; 
  outr[11] += -1.224744871391589*Ghat[3]*dv10r; 
  outr[12] += 0.7071067811865475*Ghat[8]*dv10r; 
  outr[13] += 0.7071067811865475*Ghat[9]*dv10r; 
  outr[14] += -1.224744871391589*Ghat[4]*dv10r; 
  outr[15] += 0.7071067811865475*Ghat[10]*dv10r; 
  outr[16] += -1.224744871391589*Ghat[5]*dv10r; 
  outr[17] += 0.7071067811865475*Ghat[11]*dv10r; 
  outr[18] += -1.224744871391589*Ghat[6]*dv10r; 
  outr[19] += -1.224744871391589*Ghat[7]*dv10r; 
  outr[20] += 0.7071067811865475*Ghat[12]*dv10r; 
  outr[21] += -1.224744871391589*Ghat[8]*dv10r; 
  outr[22] += -1.224744871391589*Ghat[9]*dv10r; 
  outr[23] += 0.7071067811865475*Ghat[13]*dv10r; 
  outr[24] += 0.7071067811865475*Ghat[14]*dv10r; 
  outr[25] += -1.224744871391589*Ghat[10]*dv10r; 
  outr[26] += -1.224744871391589*Ghat[11]*dv10r; 
  outr[27] += -1.224744871391589*Ghat[12]*dv10r; 
  outr[28] += 0.7071067811865475*Ghat[15]*dv10r; 
  outr[29] += -1.224744871391589*Ghat[13]*dv10r; 
  outr[30] += -1.224744871391589*Ghat[14]*dv10r; 
  outr[31] += -1.224744871391589*Ghat[15]*dv10r; 

  outl[0] += -0.7071067811865475*Ghat[0]*dv10l; 
  outl[1] += -0.7071067811865475*Ghat[1]*dv10l; 
  outl[2] += -0.7071067811865475*Ghat[2]*dv10l; 
  outl[3] += -1.224744871391589*Ghat[0]*dv10l; 
  outl[4] += -0.7071067811865475*Ghat[3]*dv10l; 
  outl[5] += -0.7071067811865475*Ghat[4]*dv10l; 
  outl[6] += -0.7071067811865475*Ghat[5]*dv10l; 
  outl[7] += -1.224744871391589*Ghat[1]*dv10l; 
  outl[8] += -1.224744871391589*Ghat[2]*dv10l; 
  outl[9] += -0.7071067811865475*Ghat[6]*dv10l; 
  outl[10] += -0.7071067811865475*Ghat[7]*dv10l; 
  outl[11] += -1.224744871391589*Ghat[3]*dv10l; 
  outl[12] += -0.7071067811865475*Ghat[8]*dv10l; 
  outl[13] += -0.7071067811865475*Ghat[9]*dv10l; 
  outl[14] += -1.224744871391589*Ghat[4]*dv10l; 
  outl[15] += -0.7071067811865475*Ghat[10]*dv10l; 
  outl[16] += -1.224744871391589*Ghat[5]*dv10l; 
  outl[17] += -0.7071067811865475*Ghat[11]*dv10l; 
  outl[18] += -1.224744871391589*Ghat[6]*dv10l; 
  outl[19] += -1.224744871391589*Ghat[7]*dv10l; 
  outl[20] += -0.7071067811865475*Ghat[12]*dv10l; 
  outl[21] += -1.224744871391589*Ghat[8]*dv10l; 
  outl[22] += -1.224744871391589*Ghat[9]*dv10l; 
  outl[23] += -0.7071067811865475*Ghat[13]*dv10l; 
  outl[24] += -0.7071067811865475*Ghat[14]*dv10l; 
  outl[25] += -1.224744871391589*Ghat[10]*dv10l; 
  outl[26] += -1.224744871391589*Ghat[11]*dv10l; 
  outl[27] += -1.224744871391589*Ghat[12]*dv10l; 
  outl[28] += -0.7071067811865475*Ghat[15]*dv10l; 
  outl[29] += -1.224744871391589*Ghat[13]*dv10l; 
  outl[30] += -1.224744871391589*Ghat[14]*dv10l; 
  outl[31] += -1.224744871391589*Ghat[15]*dv10l; 

  return std::abs(amid); 
} 
